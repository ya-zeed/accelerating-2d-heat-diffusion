#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

// Define constants
#define A 110.0
#define LENGTH 1000.0
#define TIME 80.0
#define NODES 800

// Initialize the temperature matrix and set boundary conditions on host
void initialize(double *u, int nodes) {
    int i, j;
    double step = 100.0 / (nodes - 1);

    // Set boundary conditions
    for (i = 0; i < nodes; i++) {
        double boundary_value = step * i;
        for (j = 0; j < nodes; j++) {
            u[i * nodes + j] = 20.0; // Default initialization
            if (i == 0 || i == nodes - 1) u[i * nodes + j] = boundary_value;
            if (j == 0 || j == nodes - 1) u[i * nodes + j] = boundary_value;
        }
    }
}

// Kernel to copy u into w
__global__ void copy_kernel(double *u, double *w, int nodes) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nodes && j < nodes) {
        w[i * nodes + j] = u[i * nodes + j];
    }
}

// Kernel to perform one time-step of the finite difference computation
__global__ void step_kernel(double *u, double *w, int nodes, double dx, double dy, double dt, double a) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i > 0 && i < nodes - 1 && j > 0 && j < nodes - 1) {
        double dd_ux = (w[(i - 1) * nodes + j] - 2.0 * w[i * nodes + j] + w[(i + 1) * nodes + j]) / (dx * dx);
        double dd_uy = (w[i * nodes + (j - 1)] - 2.0 * w[i * nodes + j] + w[i * nodes + (j + 1)]) / (dy * dy);
        u[i * nodes + j] = dt * a * (dd_ux + dd_uy) + w[i * nodes + j];
    }
}

int main() {
    int nodes = NODES;
    double length = LENGTH;
    double time = TIME;
    double dx = length / (nodes - 1);
    double dy = length / (nodes - 1);
    double dt = fmin(dx * dx / (4.0 * A), dy * dy / (4.0 * A));
    int t_nodes = (int)(time / dt) + 1;

    // Allocate memory on host
    double *h_u = (double *)malloc(nodes * nodes * sizeof(double));

    // Initialize on host
    initialize(h_u, nodes);

    // Allocate memory on device
    double *d_u, *d_w;
    hipMalloc((void**)&d_u, nodes * nodes * sizeof(double));
    hipMalloc((void**)&d_w, nodes * nodes * sizeof(double));

    // Copy u to device
    hipMemcpy(d_u, h_u, nodes * nodes * sizeof(double), hipMemcpyHostToDevice);

    // Set up CUDA grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((nodes + blockDim.x - 1) / blockDim.x, (nodes + blockDim.y - 1) / blockDim.y);

    // Start timer
    struct timeval start, end;
    gettimeofday(&start, NULL);

    // Time-stepping loop on host
    for (int t = 0; t < t_nodes; t++) {
        // Copy u into w
        copy_kernel<<<gridDim, blockDim>>>(d_u, d_w, nodes);

        // Compute the next timestep
        step_kernel<<<gridDim, blockDim>>>(d_u, d_w, nodes, dx, dy, dt, A);

        // Synchronize to ensure kernels have completed
        hipDeviceSynchronize();
    }

    // Stop timer
    gettimeofday(&end, NULL);

    // Copy result back to host
    hipMemcpy(h_u, d_u, nodes * nodes * sizeof(double), hipMemcpyDeviceToHost);

    // Calculate elapsed time in seconds
    double elapsed = (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 1e6;
    printf("Simulation completed in %f seconds.\n", elapsed);

    // Free memory
    free(h_u);
    hipFree(d_u);
    hipFree(d_w);

    return 0;
}
